#include <vector>

#include "caffe/layers/two_instance_embed_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void TwoInstanceEmbedDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = NULL;
  if(repeat_inputs_==0 || num_batches_so_far_==0){
    batch=this->prefetch_full_.pop("Data layer prefetch queue empty");
  } else {
    num_batches_so_far_++;
    if(num_batches_so_far_==repeat_inputs_) num_batches_so_far_=0;
  }
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  if(top.size()>=3){
    top[2]->ReshapeLike(*(batch->multilabel_[0]));
    caffe_copy(batch->multilabel_[0]->count(),
               batch->multilabel_[0]->gpu_data(),
               top[2]->mutable_gpu_data());
  }

  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  if(repeat_inputs_==0 || num_batches_so_far_==0){
    this->prefetch_free_.push(batch);
  }
}

INSTANTIATE_LAYER_GPU_FORWARD(TwoInstanceEmbedDataLayer);

}  // namespace caffe
