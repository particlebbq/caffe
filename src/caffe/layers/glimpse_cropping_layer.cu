#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/glimpse_cropping_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void GlimpseCropForward(const int nthreads,
    const Dtype* const bottom_data, const Dtype* location_tuple, 
    const int num, const int channels, const Dtype* embed_bounds, 
    const int height, const int width, const int crop_size, 
    const bool do_downsamp, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    const int iN=index/channels;
    const int ich=index%channels;

    int W=width;
    int H=height;
    if(embed_bounds!=NULL){
      H=embed_bounds[2*iN];
      W=embed_bounds[2*iN+1];
    }

    float crop_x=location_tuple[iN*2];  
    float crop_y=location_tuple[iN*2+1];  
    int crop_xmin_pix=(int)(crop_x*(H-crop_size));
    int crop_ymin_pix=(int)(crop_y*(W-crop_size));

    for(int iX=0;iX<crop_size;iX++){
      for(int iY=0;iY<crop_size;iY++){
        if(do_downsamp){
            top_data[(((iN*channels*2)+ich)*crop_size+iX)*crop_size+iY]=
                bottom_data[(((iN*channels)+ich)*height+crop_xmin_pix+iX)
                            *width+crop_ymin_pix+iY];

            int crop_xmin_pix_ds=(int)(crop_x*(H-2*crop_size));
            int crop_ymin_pix_ds=(int)(crop_y*(W-2*crop_size));

            Dtype downsamp=
              (bottom_data[(((iN*channels)+ich)*height+crop_xmin_pix_ds+iX*2)
                           *width+crop_ymin_pix_ds+iY*2]
              +bottom_data[(((iN*channels)+ich)*height+crop_xmin_pix_ds+iX*2+1)
                           *width+crop_ymin_pix_ds+iY*2]
              +bottom_data[(((iN*channels)+ich)*height+crop_xmin_pix_ds+iX*2)
                           *width+crop_ymin_pix_ds+iY*2+1]
              +bottom_data[(((iN*channels)+ich)*height+crop_xmin_pix_ds+iX*2+1)
                           *width+crop_ymin_pix_ds+iY*2+1])/4.;

            int topoff=(((iN*channels*2)+ich+channels)*crop_size+iX)*crop_size+iY;
            top_data[topoff]=downsamp;
        
        } else {
            top_data[(((iN*channels)+ich)*crop_size+iX)*crop_size+iY]=
              bottom_data[(((iN*channels)+ich)*height+crop_xmin_pix+iX)
                          *width+crop_ymin_pix+iY];
        }
      }
    }
  }
}

template <typename Dtype>
void GlimpseCroppingLayer<Dtype>::Forward_gpu(
      const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  const Dtype* full_data = bottom[0]->gpu_data();
  const Dtype* embed_bounds = NULL;
  if(bottom.size()==3){
    embed_bounds=bottom[2]->gpu_data();
  }

  const Dtype* location_tuple = bottom[1]->gpu_data();

  Dtype* glimpse_data = top[0]->mutable_gpu_data();


  GlimpseCropForward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->num()*channels_), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->num()*channels_, full_data, location_tuple, bottom[0]->num(), 
      channels_, embed_bounds,height_, width_, crop_size_, do_downsamp_, 
      glimpse_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void GlimpseCroppingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {}


INSTANTIATE_LAYER_GPU_FUNCS(GlimpseCroppingLayer);


}  // namespace caffe
