#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/gaussian_sample_layer.hpp"

namespace caffe {


template <typename Dtype>
__global__ void GaussianSampleBackward(const int nthreads, 
    const Dtype* mu, const Dtype* sigma, const Dtype* samp_diff, 
    const Dtype* sample, const int D, const int N, const bool have_dsigma, Dtype* dmu, Dtype* dsigma) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/D;
    int idim   = index%D;

    dmu[item*D+idim]=samp_diff[item*D+idim];
    if(have_dsigma){
      Dtype sig=fmaxf(0.01,fabs(sigma[item*D+idim]));
      dsigma[item*D+idim]=samp_diff[item*D+idim]
                         *(sample[item*D+idim]-mu[item*D+idim])/sig;

    }

  }

}




template <typename Dtype>
__global__ void GaussClipGrad(const int nthreads, const int D,
    const int mu_idx, const int sigma_idx, const Dtype maxval, 
    const Dtype* in_mu, const Dtype* in_sigma, Dtype* out_mu, 
    Dtype* out_sigma) {
  CUDA_KERNEL_LOOP(index, nthreads) {


    if(index>=nthreads) return;

    Dtype gradmax=fmaxf(fabs(in_mu[mu_idx]),fabs(in_sigma[sigma_idx]));
    Dtype scalefac=1.;
    if(gradmax>maxval) scalefac=maxval/gradmax;

    out_mu[index]=scalefac*in_mu[index];
    out_sigma[index]=scalefac*in_sigma[index];

  }
}




template <typename Dtype>
void GaussianSampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {


  const Dtype* mu=bottom[0]->gpu_data();
  const Dtype* sigma=NULL;
  if(bottom.size()>1) sigma=bottom[1]->gpu_data();
  Dtype* sample=top[0]->mutable_gpu_data();

  if(sigma){
    caffe_gpu_rng_gaussian<Dtype>(bottom[0]->count(),Dtype(0.),Dtype(1.),sample);
    caffe_gpu_mul<Dtype>(bottom[0]->count(),sample,sigma,sample);
  } else {
    caffe_gpu_rng_gaussian<Dtype>(bottom[0]->count(),Dtype(0.),Dtype(sigma_),sample);
  }
  caffe_gpu_add<Dtype>(bottom[0]->count(),sample,mu,sample);

  CUDA_POST_KERNEL_CHECK;


  if(top.size()>=2){
    if(bottom.size()>1){
      caffe_copy(bottom[1]->count(),
                 bottom[1]->gpu_data(),
                 top[1]->mutable_gpu_data());
    } else {
      caffe_set(top[1]->count(),
                sigma_,
                top[1]->mutable_cpu_data());
    }
  }


}


template <typename Dtype>
void GaussianSampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  const Dtype* mu=bottom[0]->gpu_data();
  const Dtype* sigma=NULL;
  if(bottom.size()>1) sigma=bottom[1]->gpu_data();
  const Dtype* sample=top[0]->gpu_data();
  const Dtype* samp_diff=top[0]->gpu_diff();
  Dtype* dmu=bottom[0]->mutable_gpu_diff();
  Dtype* dsigma=NULL;
  if(bottom.size()>1) dsigma=bottom[1]->mutable_gpu_diff();

  int N=bottom[0]->shape(0);
  int D=bottom[0]->shape(1);

  GaussianSampleBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->count(), mu, sigma, samp_diff, sample, D, N, dsigma!=NULL,
      temp_.mutable_gpu_data(), temp_.mutable_gpu_diff());  //last args are dmu, dsigma

  CUDA_POST_KERNEL_CHECK;

  if(sigma==NULL) return;

  if(cliplimit_>0){
    int maxgrad_mu_idx,maxgrad_sigma_idx;
    caffe_gpu_absmax<Dtype>(temp_.count(),temp_.gpu_data(),&maxgrad_mu_idx);
    caffe_gpu_absmax<Dtype>(temp_.count(),temp_.gpu_diff(),&maxgrad_sigma_idx);
    maxgrad_mu_idx-=1;    //correct for fortran-style indexing 
    maxgrad_sigma_idx-=1; 
  
    GaussClipGrad<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), D, maxgrad_mu_idx, maxgrad_sigma_idx, cliplimit_,
        temp_.gpu_data(), temp_.gpu_diff(), dmu, dsigma); 

    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(GaussianSampleLayer);


}  // namespace caffe
