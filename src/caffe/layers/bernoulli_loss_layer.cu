#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/bernoulli_loss_layer.hpp"

namespace caffe {



template <typename Dtype>
__global__ void BernoulliLossForward(const int nthreads,
    const Dtype* x, const Dtype* y, const int D, const int N,
    Dtype* loss_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/D;
    int idim   = index%D;

    loss_data[item*D+idim]+=(x[item*D+idim]*log(fmaxf(0.01,y[item*D+idim]))
                          +(1-x[item*D+idim])*log(fmaxf(0.01,1-y[item*D+idim])))/N;


  }
}


template <typename Dtype>
__global__ void BernoulliLossBackward(const int nthreads,
    const Dtype* x, const Dtype* y, const int D, const int N, const Dtype* loss_wt,
    Dtype* dy) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/D;
    int idim   = index%D;

    dy[item*D+idim]=-1*(x[item*D+idim]/fmaxf(0.01,y[item*D+idim])
                    -(1-x[item*D+idim])/(fmaxf(0.01,1-y[item*D+idim])))/N;

  }

}


template <typename Dtype>
__global__ void BernoulliClipGrad(const int nthreads,const int D, const Dtype* loss_wt,
    const int idx, const Dtype maxval, const Dtype* in_data,
    const Dtype* breakdown, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/D;

    Dtype scalefac=1.;
    if(breakdown){
      scalefac*=breakdown[item];
    } else {
      scalefac*=loss_wt[0];
    }
    Dtype datmax=fabs(in_data[idx]*scalefac);
    if(datmax>maxval) scalefac*=maxval/fabs(datmax);

    out_data[index]=scalefac*in_data[index];

  }
}




template <typename Dtype>
void BernoulliLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {


  const Dtype* y=bottom[0]->gpu_data();
  const Dtype* x=bottom[1]->gpu_data();
  Dtype* loss=top[0]->mutable_cpu_data();
  Dtype* breakdown=NULL;
  if(top.size()>=2){
    breakdown=top[1]->mutable_cpu_data();
  }
  int N=bottom[0]->shape(0);
  int D=bottom[0]->count(1);

  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();


    BernoulliLossForward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), x, y, D, N, loss_data);

  CUDA_POST_KERNEL_CHECK;


  int nthreads=bottom[0]->count();
  caffe_gpu_asum<Dtype>(nthreads, loss_data, loss);
  if(breakdown){
    for(int i=0;i<N;i++){
      caffe_gpu_asum(D,loss_data+i*D,breakdown+i);
    }
  }

  CUDA_POST_KERNEL_CHECK;

}


template <typename Dtype>
void BernoulliLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  const Dtype* y=bottom[0]->gpu_data();
  const Dtype* x=bottom[1]->gpu_data();
  const Dtype* loss_wt=top[0]->gpu_diff();

  const Dtype* breakdown=NULL;
  if(top.size()>=2) {
    breakdown=top[1]->gpu_diff();
  }

  Dtype* dy=bottom[0]->mutable_gpu_diff();


  int N=bottom[0]->shape(0);
  int D=bottom[0]->shape(1);

    BernoulliLossBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), bottom[1]->gpu_data(), bottom[0]->gpu_data(), D, N, loss_wt, temp_.mutable_gpu_diff()); 


  if(cliplimit_>0){
    int maxidx;
    caffe_gpu_absmax<Dtype>(temp_.count(),temp_.gpu_diff(),&maxidx);
    maxidx-=1;  //because the returned value is in fortran-style indexing

    BernoulliClipGrad<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), D, loss_wt, maxidx, cliplimit_, temp_.gpu_diff(), breakdown, bottom[0]->mutable_gpu_diff());  //dy

    CUDA_POST_KERNEL_CHECK;
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(BernoulliLossLayer);


}  // namespace caffe
