#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/bernoulli_sample_layer.hpp"

namespace caffe {



template <typename Dtype>
__global__ void BernoulliSampleForward(const int nthreads, const Dtype* p, 
      const int D, Dtype* sample){

  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/D;
    int idim   = index%D;

    if(sample[item*D+idim]>p[item*D+idim]) {
      sample[item*D+idim]=0;
    } else {
      sample[item*D+idim]=1;
    }
  }

}


template <typename Dtype>
void BernoulliSampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {


  const Dtype* p=bottom[0]->gpu_data();
  Dtype* sample=top[0]->mutable_gpu_data();
  int N=bottom[0]->shape(0);
  int D=bottom[0]->count(1);

  caffe_gpu_rng_uniform<Dtype>(bottom[0]->count(),0.,1.,sample);

    BernoulliSampleForward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), p, D, sample);


  CUDA_POST_KERNEL_CHECK;


}


template <typename Dtype>
void BernoulliSampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

}

INSTANTIATE_LAYER_GPU_FUNCS(BernoulliSampleLayer);


}  // namespace caffe
