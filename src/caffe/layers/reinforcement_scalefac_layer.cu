#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/reinforcement_scalefac_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReinforcementScalefacBackward(const int nthreads,
    const Dtype* in_diff, const Dtype* R, const Dtype* b, const int C, const Dtype* curr, const Dtype lambda, 
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/C;
    int ic   = index%C;

    Dtype curr_weight=1.;

    if(curr) curr_weight=curr[item];
    out_diff[item*C+ic]=lambda*curr_weight*(R[item]-b[item])*in_diff[item*C+ic];

  }

}

template <typename Dtype>
void ReinforcementScalefacLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  caffe_copy(bottom[0]->count(),
             bottom[0]->gpu_data(),
             top[0]->mutable_gpu_data());
}


template <typename Dtype>
void ReinforcementScalefacLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {


  Dtype* out_diff=bottom[0]->mutable_gpu_diff();
  const Dtype* in_diff=top[0]->gpu_diff();
  const Dtype* b=bottom[1]->gpu_data();
  const Dtype* R=top[1]->gpu_diff();
  const Dtype* curr=NULL;
  if(top.size()>2) curr=top[2]->gpu_diff();
  int C=bottom[0]->count(1);

  ReinforcementScalefacBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->count(), in_diff, R, b, C, curr, lambda_, out_diff);

  CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(ReinforcementScalefacLayer);


}  // namespace caffe
