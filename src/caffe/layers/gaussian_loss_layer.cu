#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/gaussian_loss_layer.hpp"

namespace caffe {



template <typename Dtype>
__global__ void GaussianLossForward(const int nthreads,
    const Dtype* x, const Dtype* mu, const Dtype* sigma, const int D, 
    const int N, const int bottom_size, Dtype* loss_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/D;
    int idim   = index%D;

    Dtype logprob=0;
    Dtype sig=fmaxf(0.01,fabs(sigma[item*D+idim]));
    if(bottom_size>2){
      logprob-=-log(sqrt(2*3.14159265*sig*sig))
              -(pow(x[item*D+idim]-mu[item*D+idim],2)/(2*pow(sig,2)) );
    } else {
      logprob-=0.5*(1+log(sig*sig)
                        -(mu[item*D+idim]*mu[item*D+idim])
                        -(sig*sig));
    }
    loss_data[index]=logprob/N;
  }

}



template <typename Dtype>
__global__ void GaussianLossBackward(const int nthreads, const Dtype* x, 
    const Dtype* mu, const Dtype* sigma, const int D, const int N, 
    const int bottom_size, Dtype* dmu, Dtype* dsigma) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    if(index>=nthreads) return;

    int item = index/D;
    int idim   = index%D;

    Dtype sig=fmaxf(0.01,fabs(sigma[item*D+idim]));
    if(bottom_size>2){
      dmu[item*D+idim]=(-1*(x[item*D+idim]-mu[item*D+idim])/(sig*sig))/N;
      dsigma[item*D+idim]=
         (1/sig - pow(x[item*D+idim]-mu[item*D+idim],2)/pow(sig,3))/N;

    } else {

      dmu[item*D+idim]=mu[item*D+idim]/N;
      dsigma[item*D+idim]=(-1*(1/sig - sig))/N;

    }

  }

}




template <typename Dtype>
__global__ void GaussClipGrad(const int nthreads, const int D,
    const Dtype* loss_wt, const int mu_idx, const int sigma_idx, 
    const Dtype maxval, const Dtype* in_mu, const Dtype* in_sigma, 
    const Dtype* breakdown, Dtype* out_mu, Dtype* out_sigma) {
  CUDA_KERNEL_LOOP(index, nthreads) {


    if(index>=nthreads) return;

    int item=index/D;

    Dtype gradmax=fmaxf(fabs(in_mu[mu_idx]),fabs(in_sigma[sigma_idx]));
    Dtype scalefac=1.;
    if(maxval>0 && gradmax>maxval) scalefac=maxval/gradmax;
    if(breakdown){
      scalefac*=breakdown[item];
    } else {
      scalefac*=loss_wt[0];
    }

    out_mu[index]=scalefac*in_mu[index];
    out_sigma[index]=scalefac*in_sigma[index];

  }
}




template <typename Dtype>
void GaussianLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {


  const Dtype* mu=bottom[0]->gpu_data();
  const Dtype* sigma=bottom[1]->gpu_data();
  const Dtype* x=NULL;
  if(bottom.size()>2){
    x=bottom[2]->gpu_data();
  }
  Dtype* loss=top[0]->mutable_cpu_data();
  Dtype* breakdown=NULL;
  if(top.size()>=2){
    breakdown=top[1]->mutable_cpu_data();
  }


  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();

  int N=bottom[0]->shape(0);
  int D=bottom[0]->count(1);

  loss[0]=0;

    GaussianLossForward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), x, mu, sigma, D, N, bottom.size(), loss_data);

  int nthreads=bottom[0]->count();
  caffe_gpu_asum<Dtype>(nthreads, loss_data, loss);
  if(breakdown){
    for(int i=0;i<N;i++){
      caffe_gpu_asum(D,loss_data+i*D,breakdown+i);
    }
  }

  CUDA_POST_KERNEL_CHECK;

}


template <typename Dtype>
void GaussianLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {


  const Dtype* mu=bottom[0]->gpu_data();
  const Dtype* sigma=bottom[1]->gpu_data();
  const Dtype* x=NULL;
  if(bottom.size()>2){
    x=bottom[2]->gpu_data();
  }
  const Dtype* loss_wt=top[0]->gpu_diff();
  const Dtype* breakdown=NULL;
  if(top.size()>=2) {
    breakdown=top[1]->gpu_diff();
  }

  Dtype* dmu=bottom[0]->mutable_gpu_diff();
  Dtype* dsigma=bottom[1]->mutable_gpu_diff();

  int N=bottom[0]->shape(0);
  int D=bottom[0]->shape(1);


  if(breakdown){  //we will apply loss weights from breakdown, not global loss, during clip grad later
    GaussianLossBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), x, mu, sigma, D, N, bottom.size(), 
        temp_.mutable_gpu_data(), temp_.mutable_gpu_diff());  //last args are dmu, dsigma
  } else {
    GaussianLossBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), x, mu, sigma, D, N, bottom.size(),
        temp_.mutable_gpu_data(), temp_.mutable_gpu_diff());  //last args are dmu, dsigma
  }

  int maxgrad_mu_idx,maxgrad_sigma_idx;
  caffe_gpu_absmax<Dtype>(temp_.count(),temp_.gpu_data(),&maxgrad_mu_idx);
  caffe_gpu_absmax<Dtype>(temp_.count(),temp_.gpu_diff(),&maxgrad_sigma_idx);
  maxgrad_mu_idx-=1;    //correct for fortran-style indexing 
  maxgrad_sigma_idx-=1; 

  GaussClipGrad<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->count(), D, loss_wt, maxgrad_mu_idx, maxgrad_sigma_idx, 
      cliplimit_,temp_.gpu_data(), temp_.gpu_diff(), breakdown,
      bottom[0]->mutable_gpu_diff(), bottom[1]->mutable_gpu_diff());  //dmu, dsigma


  CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(GaussianLossLayer);


}  // namespace caffe
